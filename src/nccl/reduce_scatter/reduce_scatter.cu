#include "hip/hip_runtime.h"
// filepath: src/nccl/reduce_scatter/reduce_scatter.cu
#include "../common/nccl_context.hpp"
#include "../../common/include/arg_parser.hpp"
#include "../../common/include/logger.hpp"
#include <chrono>
#include <iostream>
#include <iomanip>
#include <string>

// Kernel to initialize device buffers for reduce_scatter
template<typename T>
__global__ void init_buffers(T* send_buf, T* recv_buf, size_t count, int size, int rank) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = count * size;
    if (id < total) {
        // Data pattern: unique per rank
        send_buf[id] = static_cast<T>((rank + 1) * (id + 1));
    }
    if (id < count) {
        // initialize receive buffer to sentinel
        recv_buf[id] = static_cast<T>(-1);
    }
}

// Run NCCL reduce_scatter for type T
template<typename T>
void run_reduce_scatter(size_t count, int size, int rank, NcclContext& ctx, const std::string& data_type) {
    // determine NCCL data type
    ncclDataType_t nccl_dtype;
    if (data_type == "int") nccl_dtype = ncclInt;
    else if (data_type == "float") nccl_dtype = ncclFloat;
    else /* double */ nccl_dtype = ncclDouble;

    // allocate device buffers
    T* send_buf;
    T* recv_buf;
    hipMalloc(&send_buf, count * size * sizeof(T));
    hipMalloc(&recv_buf, count * sizeof(T));

    // initialize buffers
    int threads = 256;
    int blocks = (count * size + threads - 1) / threads;
    init_buffers<T><<<blocks, threads, 0, ctx.stream>>>(send_buf, recv_buf, count, size, rank);
    hipStreamSynchronize(ctx.stream);

    // warm-up non misurata
    ncclReduceScatter(send_buf, recv_buf, count, nccl_dtype, ncclSum, ctx.comm, ctx.stream);
    hipStreamSynchronize(ctx.stream);

    // perform reduce_scatter and time it 5 times
    for (int iter = 0; iter < 5; ++iter) {
        auto t_start = std::chrono::high_resolution_clock::now();
        ncclReduceScatter(send_buf, recv_buf, count, nccl_dtype, ncclSum, ctx.comm, ctx.stream);
        hipStreamSynchronize(ctx.stream);
        auto t_end = std::chrono::high_resolution_clock::now();
        double elapsed_ms = std::chrono::duration_cast<std::chrono::microseconds>(t_end - t_start).count() / 1000.0;
        ctx.logger.log_result_with_gdr_detection(data_type, count, size, rank, elapsed_ms);
        std::cout << "Rank " << rank << " reduce_scatter time (iter " << iter << "): "
                  << std::fixed << std::setprecision(3) << elapsed_ms << " ms\n";
    }

    // correctness check
    T* host_buf = new T[count];
    hipMemcpy(host_buf, recv_buf, count * sizeof(T), hipMemcpyDeviceToHost);
    bool ok = true;
    for (size_t i = 0; i < count && ok; ++i) {
        size_t global_idx = rank * count + i;
        T expected = static_cast<T>(0);
        for (int r = 0; r < size; ++r) {
            expected += static_cast<T>((r + 1) * (global_idx + 1));
        }
        if (host_buf[i] != expected) {
            ok = false;
        }
    }
    std::cout << (ok ? "PASSED\n" : "FAILED\n");
    delete[] host_buf;

    // cleanup
    hipFree(send_buf);
    hipFree(recv_buf);
}

int main(int argc, char* argv[]) {
    ArgParser parser(argc, argv);
    parser.add<std::string>("--dtype").add<size_t>("--count");
    parser.parse();

    std::string dtype = parser.get<std::string>("--dtype");
    size_t count = parser.get<size_t>("--count");
    std::string output_dir;
    try {
        output_dir = parser.get<std::string>("--output");
    } catch (...) {
        output_dir = "";
    }
    if (count == 0) count = 1024 * 1024;

    // Initialize NCCL context for reduce_scatter
    auto ctx = init_nccl(output_dir, "reduce_scatter", argc, argv);
    int size = ctx.size;
    int rank = ctx.rank;

    // dispatch based on data type
    if (dtype == "int") {
        run_reduce_scatter<int>(count, size, rank, ctx, dtype);
    } else if (dtype == "float") {
        run_reduce_scatter<float>(count, size, rank, ctx, dtype);
    } else if (dtype == "double") {
        run_reduce_scatter<double>(count, size, rank, ctx, dtype);
    } else {
        std::cerr << "Unsupported dtype: " << dtype << std::endl;
        return -1;
    }
    return 0;
}
