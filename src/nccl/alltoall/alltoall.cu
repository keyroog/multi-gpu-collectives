#include "hip/hip_runtime.h"
// filepath: src/nccl/alltoall/alltoall.cu
#include "../common/nccl_context.hpp"
#include "../../common/include/arg_parser.hpp"
#include "../../common/include/logger.hpp"
#include <chrono>
#include <iostream>
#include <iomanip>
#include <string>

// Kernel to initialize device buffers for alltoall
template<typename T>
__global__ void init_buffers(T* send_buf, T* recv_buf, size_t count, int rank, int size) {
    size_t id = blockIdx.x * blockDim.x + threadIdx.x;
    size_t total = count * size;
    if (id < total) {
        int dest_rank = id / count;
        size_t local_id = id % count;
        send_buf[id] = static_cast<T>(rank * 1000 + dest_rank * 100 + local_id);
        recv_buf[id] = static_cast<T>(-1);
    }
}

// Run NCCL alltoall for type T using point-to-point sends/receives
template<typename T>
void run_alltoall(size_t count, int size, int rank, NcclContext& ctx, const std::string& data_type) {
    // determine NCCL data type
    ncclDataType_t nccl_dtype;
    if (data_type == "int") nccl_dtype = ncclInt;
    else if (data_type == "float") nccl_dtype = ncclFloat;
    else /* double */ nccl_dtype = ncclDouble;

    // allocate device buffers
    T* send_buf;
    T* recv_buf;
    hipMalloc(&send_buf, count * size * sizeof(T));
    hipMalloc(&recv_buf, count * size * sizeof(T));

    // initialize buffers
    int threads = 256;
    int blocks = (count * size + threads - 1) / threads;
    init_buffers<T><<<blocks, threads, 0, ctx.stream>>>(send_buf, recv_buf, count, rank, size);
    hipStreamSynchronize(ctx.stream);

    // perform alltoall and time it 5 times
    for (int iter = 0; iter < 5; ++iter) {
        auto t_start = std::chrono::high_resolution_clock::now();
        ncclGroupStart();
        for (int peer = 0; peer < size; ++peer) {
            ncclRecv(recv_buf + peer * count, count, nccl_dtype, peer, ctx.comm, ctx.stream);
            ncclSend(send_buf + peer * count, count, nccl_dtype, peer, ctx.comm, ctx.stream);
        }
        ncclGroupEnd();
        hipStreamSynchronize(ctx.stream);
        auto t_end = std::chrono::high_resolution_clock::now();
        double elapsed_ms = std::chrono::duration_cast<std::chrono::microseconds>(t_end - t_start).count() / 1000.0;
        ctx.logger.log_result_with_gdr_detection(data_type, count, size, rank, elapsed_ms);
        std::cout << "Rank " << rank << " alltoall time (iter " << iter << "): "
                  << std::fixed << std::setprecision(3) << elapsed_ms << " ms\n";
    }

    // correctness check
    T* host_buf = new T[count * size];
    hipMemcpy(host_buf, recv_buf, count * size * sizeof(T), hipMemcpyDeviceToHost);
    bool ok = true;
    for (int src = 0; src < size && ok; ++src) {
        for (size_t i = 0; i < count; ++i) {
            T expected = static_cast<T>(src * 1000 + rank * 100 + i);
            if (host_buf[src * count + i] != expected) { ok = false; break; }
        }
    }
    std::cout << (ok ? "PASSED\n" : "FAILED\n");
    delete[] host_buf;

    hipFree(send_buf);
    hipFree(recv_buf);
}

int main(int argc, char* argv[]) {
    ArgParser parser(argc, argv);
    parser.add<std::string>("--dtype").add<size_t>("--count");
    parser.parse();

    std::string dtype = parser.get<std::string>("--dtype");
    size_t count = parser.get<size_t>("--count");
    std::string output_dir;
    try {
        output_dir = parser.get<std::string>("--output");
    } catch (...) {
        output_dir = "";
    }
    if (count == 0) count = 1024 * 1024;

    // Initialize NCCL context for alltoall
    auto ctx = init_nccl(output_dir, "alltoall", argc, argv);
    int size = ctx.size;
    int rank = ctx.rank;

    // dispatch based on data type
    if (dtype == "int") {
        run_alltoall<int>(count, size, rank, ctx, dtype);
    } else if (dtype == "float") {
        run_alltoall<float>(count, size, rank, ctx, dtype);
    } else if (dtype == "double") {
        run_alltoall<double>(count, size, rank, ctx, dtype);
    } else {
        std::cerr << "Unsupported dtype: " << dtype << std::endl;
        return -1;
    }
    return 0;
}
